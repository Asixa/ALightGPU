#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdlib.h"
#include <thread>

#include "header/Hitable.h"
#include "header/Camera.h"
#include "header/GLWindow.h"
#include "hiprand/hiprand_kernel.h"
using namespace std;

Camera cam;
GPUHitable *gpu_world;
void InitData()
{
	PixelLength = ImageHeight * ImageWidth * 4;
	PixelData = new GLbyte[PixelLength];
	for (auto i = 0; i < PixelLength; i++)
		PixelData[i] = static_cast<GLbyte>(int(0));
}

__device__ bool HitTest(GPUHitable* list, int size, Ray r, float t_min, float t_max,  HitRecord& rec)
{
	HitRecord temp_rec;
	auto hit_anything = false;
	double closest_so_far = t_max;
	for (auto i=0;i<size;i++)
	{
		if (list[i].Hit(r, t_min, closest_so_far, temp_rec)) {
			hit_anything = true;
			closest_so_far = temp_rec.t;
			rec = temp_rec;
		}
	}
	return hit_anything;
}

__global__ 
void Sampler(int d_width, int d_height, int worldsize, GPUHitable* d_world, byte * d_pixeldata, Camera* d_camera, hiprandState *const rngStates)
{
	// Determine thread ID
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto tid2 = blockIdx.y * blockDim.y + threadIdx.y;
	// Initialise the RNG
	const unsigned int seed = 1234;
	//***	hiprandState random = rngStates[tid];
	hiprand_init(seed, tid, tid2, &rngStates[tid]);			//��ʼ�������
	Vec3 color(0, 0, 0);


	//**********  Debug Specific Pixel **********
	//int x = 256, y = 256;
	//float u = float(x) / float(512);
	//float v = float(y) / float(512);

	const int x = blockIdx.x * 16 + threadIdx.x,y = blockIdx.y * 16 + threadIdx.y;

	for (auto j = 0; j < SPP; j++) {
		const auto u = float(x + hiprand_uniform(&rngStates[tid])) / float(512),
		v = float(y + hiprand_uniform(&rngStates[tid])) / float(512);
		Ray ray(Vec3(0, 0, 0), Vec3(-2.0, -2.0, -1.0) + u * Vec3(4, 0, 0) + v * Vec3(0, 4, 0));
		Vec3 c(0, 0, 0);
		float factor = 1;
		for (auto i = 0; i < MAX_SCATTER_TIME; i++)
		{
			HitRecord rec;
			if (HitTest(d_world, worldsize, ray, 0, 99999, rec))
			{
				// random in unit sphere
				Vec3 random_in_unit_sphere;
				do random_in_unit_sphere = 2.0*Vec3(hiprand_uniform(&rngStates[tid]), hiprand_uniform(&rngStates[tid]), hiprand_uniform(&rngStates[tid])) - Vec3(1, 1, 1);
				while (random_in_unit_sphere.squared_length() >= 1.0);

				factor /= 2;
				auto target = rec.p + rec.normal + random_in_unit_sphere;
				ray = Ray(rec.p, target - rec.p);

				//****** �����������������غ�ɫ ******
				if (i == MAX_SCATTER_TIME-1)
				{
					c = Vec3(0, 0, 0);
					break;
				}
			}
			else
			{
				const auto t = 0.5*(unit_vector(ray.Direction()).y() + 1);
				c = factor * ((1.0 - t)*Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0));
				break;
			}
		}
		color += c;
	}
	color /= SPP;

	//SetColor
	const auto i = 512 * 4 * y + x * 4;
	d_pixeldata[i] = color.r() * 255;
	d_pixeldata[i + 1] = color.g() * 255;
	d_pixeldata[i + 2] = color.b() * 255;
	d_pixeldata[i + 3] = 255;
}

hipError_t GPURender()
{
	const auto begin = clock();

	//****** ����GPU�Դ�ָ�� ******
	Camera * d_camera;
	int * d_Width = 0;
	int * d_Height = 0;
	byte * d_pixeldata;
	GPUHitable * d_world_gpu;
	const auto h_pixeldata = new byte[ImageWidth*ImageHeight * 4];
	for (auto i = 0; i < ImageWidth*ImageHeight * 4; i++)h_pixeldata[i]=byte(0);		//�������ػ����ʼֵ

	//***** Init Scene Data ******
	float p1[4] = { 0,0,-1,0.5 };
	float p2[4] = { 0,-100.5,-1,100 };
	GPUHitable w[2] = { GPUHitable(p1),GPUHitable(p2) };
	//****************************

	cout << "׼����ʼ��Ⱦ" << endl;
	const auto cuda_status = hipSetDevice(0);											// Cuda Status for checking error

	hiprandState *d_rng_states = nullptr; //�����
	dim3 grid(512 / BlockSize, 512 / BlockSize), block(BlockSize, BlockSize);			// Split area, 32*32/block
	//dim3 grid(1),block(1);  this line for debuging specific pixel

	//******  �����ַ ****** 
	hipMalloc(reinterpret_cast<void**>(&d_Width), sizeof(int));
	hipMalloc(reinterpret_cast<void**>(&d_Height), sizeof(int));
	hipMalloc(reinterpret_cast<void**>(&d_world_gpu), 24*2);
	hipMalloc(reinterpret_cast<void**>(&d_pixeldata), 512*512*4*sizeof(byte));
	hipMalloc(reinterpret_cast<void**>(&d_camera), sizeof(cam));
	hipMalloc(reinterpret_cast<void **>(&d_rng_states), grid.x * block.x * sizeof(hiprandState));

	//****** �ڴ渴�� host->Device ******
	hipMemcpy(d_Width, &ImageWidth, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Height, &ImageHeight, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_world_gpu, &w, 24*2, hipMemcpyHostToDevice);
	hipMemcpy(d_camera, &cam, sizeof(Camera), hipMemcpyHostToDevice);
	hipMemcpy(d_pixeldata, h_pixeldata, 512 * 512 * 4 * sizeof(byte), hipMemcpyHostToDevice);

	//******�����߳� ******
	Sampler <<<grid,block>>>(512,512,2,d_world_gpu,d_pixeldata,d_camera, d_rng_states);

	//****** �����ڴ� Device->host ******
	hipMemcpy(h_pixeldata, d_pixeldata, 512 * 512 * 4 * sizeof(byte), hipMemcpyDeviceToHost);

	//****** ת���������� ****** TODO �����Ż�
	for (auto i = 0; i < ImageWidth*ImageHeight * 4; i++)PixelData[i] = h_pixeldata[i];
	
	printf("��Ⱦ��ɣ�������ʱ��: %lf��", double(clock() - begin) / CLOCKS_PER_SEC);
	return cuda_status;
}


int main(int argc, char* argv[])
{
	InitData();
	GPURender();
	InitWindow(argc, argv, GLUT_DOUBLE | GLUT_RGBA, 100, 100, ImageWidth, ImageHeight, "ALightGPU");
	return 0;
}


