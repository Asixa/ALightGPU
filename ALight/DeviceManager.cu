#include "DeviceManager.h"
#include <hip/hip_runtime_api.h>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "RayTracer.h"
#include "Setting.h"
#include "RTSampler.h"
#include <iostream>
#include "Float2Byte.h"
#include "Engine.h"


DeviceManager::DeviceManager()
{
}


DeviceManager::~DeviceManager()
{
}

void DeviceManager::PrintDeviceInfo()
{
	auto device_count = 0;
	hipGetDeviceCount(&device_count);

	if (device_count == 0)
	{
		printf("û��֧��CUDA���豸!\n");
		return;
	}
	for (auto dev = 0; dev < device_count; dev++)
	{
		hipSetDevice(dev);
		hipDeviceProp_t device_prop{};
		hipGetDeviceProperties(&device_prop, dev);
		printf("�豸 %d: \"%s\"\n", dev, device_prop.name);
		char msg[256];
		sprintf_s(msg, sizeof(msg),
			"global memory��С:        %.0f MBytes "
			"(%llu bytes)\n",
			static_cast<float>(device_prop.totalGlobalMem / 1048576.0f),
			static_cast<unsigned long long>(device_prop.totalGlobalMem));
		printf("%s", msg);
		printf("SM��:                    %2d \nÿSM CUDA������:           %3d \n��CUDA������:             %d \n",
			device_prop.multiProcessorCount,
			_ConvertSMVer2Cores(device_prop.major, device_prop.minor),
			_ConvertSMVer2Cores(device_prop.major, device_prop.minor) *
			device_prop.multiProcessorCount);
		printf("��̬�ڴ��С:             %zu bytes\n",
			device_prop.totalConstMem);
		printf("ÿblock�����ڴ��С:      %zu bytes\n",
			device_prop.sharedMemPerBlock);
		printf("ÿblock�Ĵ�����:          %d\n",
			device_prop.regsPerBlock);
		printf("�߳�����С:               %d\n",
			device_prop.warpSize);
		printf("ÿ����������߳���:       %d\n",
			device_prop.maxThreadsPerMultiProcessor);
		printf("ÿblock����߳���:        %d\n",
			device_prop.maxThreadsPerBlock);
		printf("�߳̿����ά�ȴ�С        (%d, %d, %d)\n",
			device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1],
			device_prop.maxThreadsDim[2]);
		printf("�������ά�ȴ�С          (%d, %d, %d)\n",
			device_prop.maxGridSize[0], device_prop.maxGridSize[1],
			device_prop.maxGridSize[2]);
		printf("\n");
	}
	printf("************�豸��Ϣ��ӡ���************\n\n");
}

void DeviceManager::Init(RayTracer* tracer)
{
	ray_tracer = tracer;
	grid = dim3(ray_tracer->Width / Setting::BlockSize, ray_tracer->Height / Setting::BlockSize);
	block = dim3(Setting::BlockSize, Setting::BlockSize);

	host_float_data = new float[ray_tracer->Width * ray_tracer->Height * 4];
	hipMalloc(reinterpret_cast<void**>(&devicde_float_data), ray_tracer->Width * ray_tracer->Height * 4 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&devicde_byte_data), ray_tracer->Width * ray_tracer->Height * 4 * sizeof(GLbyte));
	hipMalloc(reinterpret_cast<void**>(&rng_states), grid.x * block.x * sizeof(hiprandState));
	hipMalloc(reinterpret_cast<void**>(&d_camera), sizeof(Camera));
}

void DeviceManager::Run()
{
	// printf("Hello");

	//****** �����ڴ� host->device ******
	hipMemcpy(devicde_float_data, host_float_data, ray_tracer->Width * ray_tracer->Height * 4 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_camera, Engine::Instance()->camera, sizeof(Camera), hipMemcpyHostToDevice);
	SetConstants();
	//dim3 grid(ray_tracer->Width / Setting::BlockSize, ray_tracer->Height / Setting::BlockSize), block(Setting::BlockSize, Setting::BlockSize);


	ray_tracer->Sampled += Setting::SPP;
	IPRSampler << <grid, block >> > (ray_tracer->Width, ray_tracer->Height, (rand() / (RAND_MAX + 1.0)) * 1000, Setting::SPP, ray_tracer->Sampled, 4, 0, devicde_float_data, rng_states, d_camera);
	Float2Byte <<<grid, block >> > (ray_tracer->Width, ray_tracer->Sampled, Setting::SPP, devicde_float_data, devicde_byte_data);
	
	hipDeviceSynchronize();
	
	//****** �����ڴ� Device->host ******
	hipMemcpy(host_float_data, devicde_float_data, ray_tracer->Width * ray_tracer->Height * 4 * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(ray_tracer->Data, devicde_byte_data, ray_tracer->Width * ray_tracer->Height * 4 * sizeof(GLbyte), hipMemcpyDeviceToHost);

	const auto error = hipGetLastError();
	if (error != 0)printf("Cuda Error %d\n", error);
}
