#include "hip/hip_runtime.h"
#include "RTSampler.h"
#include "Ray.h"
#include "Camera.h"
#include "float3Extension.h"
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include "Objects.h"
#include "float3x3.h"
#include "RTDeviceData.h"
#include "SurfaceHitRecord.h"
#include "BVH.h"
#include <cstdio>


__device__ Ray CreateCameraRay(Camera* camera, float u, float v)
{
	return Ray(camera->Origin, UnitVector(camera->LowerLeftCorner + u * camera->Horizontal + v * camera->Vertical - camera->Origin));
}
__device__ SurfaceHitRecord CreateRayHit()
{
	SurfaceHitRecord hit;
	hit.p = make_float3(0.0f, 0.0f, 0.0f);
	hit.t =INF;
	hit.normal = make_float3(0.0f, 0.0f, 0.0f);
	return hit;
}
__device__ bool IntersectTriangle_MT97(Ray ray, float3 vert0, float3 vert1, float3 vert2,float& t, float& u, float& v)
{
	// find vectors for two edges sharing vert0
	float3 edge1 = vert1 - vert0;
	float3 edge2 = vert2 - vert0;
	// begin calculating determinant - also used to calculate U parameter
	float3 pvec = Cross(ray.direction, edge2);
	// if determinant is near zero, ray lies in plane of triangle
	float det = Dot(edge1, pvec);
	// use backface culling
	if (det < EPSILON)return false;
	float inv_det = 1.0f / det;
	// calculate distance from vert0 to ray origin
	float3 tvec = ray.origin - vert0;
	// calculate U parameter and test bounds
	u = Dot(tvec, pvec) * inv_det;
	if (u < 0.0 || u > 1.0f)
		return false;
	// prepare to test V parameter
	float3 qvec = Cross(tvec, edge1);
	// calculate V parameter and test bounds
	v = Dot(ray.direction, qvec) * inv_det;
	if (v < 0.0 || u + v > 1.0f)
		return false;
	// calculate t, ray intersects triangle
	t = Dot(edge2, qvec) * inv_det;
	return true;
}

__device__ void GetUV(Vertex v0, Vertex v1, Vertex v2, float3 p, float3& normal,float2& uv)
{
	auto f1 = v0.point - p;
	auto f2 = v1.point - p;
	auto f3 = v2.point - p;
	//������������ӣ�����˳���޹ؽ�Ҫ����
	auto a = Length(Cross(v0.point - v1.point, v0.point - v2.point)); // ����������� a
	auto a1 = Length(Cross(f2, f3)) / a; // p1 ��������� / a
	auto a2 = Length(Cross(f3, f1)) / a; // p2 ��������� / a 
	auto a3 = Length(Cross(f1, f2)) / a; // p3 ��������� / a
	// �ҵ���Ӧ�ڵ�f��uv��uv1 / uv2 / uv3��p1 / p2 / p3��أ���
	//uv = v0.uv * a1 + v1.uv * a2 + v2.uv * a3;
	// �ҵ���Ӧ�ڵ�f�ķ��ߣ�����1 / ����2 / ����3��p1 / p2 / p3��أ���
	normal = v0.normal * a1 + v1.normal * a2 + v2.normal * a3;
}
__device__ void IntersectTriangle(Ray ray,  SurfaceHitRecord* bestHit, RTDeviceData& data,  int material,
                                  Vertex vert0, Vertex vert1, Vertex vert2)
{
	float t, u, v;
	if (IntersectTriangle_MT97(ray, vert0.point, vert1.point, vert2.point, t, u, v))
	{
		if (t > 0 && t < bestHit->t)
		{
			bestHit->t = t;
			bestHit->p = ray.origin + t * ray.direction;
			if(Length(vert0.normal)==0)bestHit->normal = UnitVector(Cross(vert1.point - vert0.point, vert2.point - vert0.point));
			else GetUV(vert0, vert1, vert2, bestHit->p, bestHit->normal, bestHit->uv);
			//bestHit->normal = UnitVector(Cross(vert1.point - vert0.point, vert2.point - vert0.point));
			bestHit->mat_ptr= &data.Materials[material];
		}
	}
}

__device__ void IntersectGroundPlane(Ray ray, SurfaceHitRecord* bestHit,RTDeviceData& data)
{
	const auto t = -ray.origin.y / ray.direction.y;
	if (t > 0.001 && t < bestHit->t)
	{
		bestHit->t = t;
		bestHit->p = ray.origin + t * ray.direction;
		bestHit->normal = make_float3(0.0f, 1.0f, 0.0f);
		bestHit->mat_ptr = &data.Materials[0];
	}
}
__device__ void IntersectSphere(Ray ray, SurfaceHitRecord* best_hit, const Sphere sphere,RTDeviceData& data,int material)
{
	const auto d = ray.origin - sphere.position;
	const auto p1 = Dot(ray.direction, d) * -1;
	const auto p2_sqr = p1 * p1 - Dot(d, d) + sphere.radius * sphere.radius;
	if (p2_sqr < 0)return;
	const auto p2 = sqrt(p2_sqr);
	const auto t = p1 - p2 > 0 ? p1 - p2 : p1 + p2;
	if (t > 0.001 && t < best_hit->t)
	{
		best_hit->t = t;
		best_hit->p = ray.origin + t * ray.direction;
		best_hit->normal = UnitVector(best_hit->p - sphere.position);
		best_hit->mat_ptr = &data.Materials[material];
	}
}
__device__ float3x3 GetTangentSpace(float3 normal)
{
	const auto helper =  (fabs(normal.x) > 0.99f)? make_float3(0, 0, 1):make_float3(1, 0, 0);
	const auto tangent = UnitVector(Cross(normal, helper));
	const auto binormal = UnitVector(Cross(normal, tangent));
	return float3x3(tangent, binormal, normal);
}
__device__ bool HitAABB(const Ray& r, AABB* aabb, float tmin = 0, float tmax = FLT_MAX)
{
	for (auto a = 0; a < 3; a++)
	{
		const auto t0 = fmin((Get(aabb->min, a) - Get(r.origin, a)) / Get(r.direction, a),
		                     (Get(aabb->max, a) - Get(r.origin, a)) / Get(r.direction, a));
		const auto t1 = fmax((Get(aabb->min, a) - Get(r.origin, a)) / Get(r.direction, a),
		                     (Get(aabb->max, a) - Get(r.origin, a)) / Get(r.direction, a));
		tmin = fmax(t0, tmin);
		tmax = fmin(t1, tmax);
		if (tmax <= tmin)return false;
	}
	return true;
}

__device__ SurfaceHitRecord Trace(Ray ray,RTDeviceData& data)
{
	auto best_hit = CreateRayHit();
	BVH* stack[20];
	int ptr = 0;
	auto current = data.bvh;
	do
	{
		auto left = current->left;
		auto right = current->right;
		if (current->tri)
		{
			IntersectTriangle(ray, &best_hit, data, current->triangle->mat,
				current->triangle->v2,
				current->triangle->v1,
				current->triangle->v3);
			if (ptr <= 0)current = nullptr;
			else  current = stack[--ptr];
		}
		else
		{
			if (HitAABB(ray, current->aabb))
			{
				current= left;
				stack[ptr++]= right;
			}
			else
			{
				if (ptr <= 0)current = nullptr;
				else  current = stack[--ptr];
				
			}
		}
	} while (current!=nullptr);

	free(stack);
	if(data.ground)IntersectGroundPlane(ray, &best_hit,data);



	// IntersectSphere(ray, &best_hit, Sphere(make_float3(0, 1, 0),1,make_float3(1,0,0),make_float3(0, 0, 0),1,make_float3(0,0,0)),data,1);
	// IntersectSphere(ray, &best_hit,Sphere(make_float3(2, 1, 0), 1, make_float3(1, 0, 0), make_float3(0, 0, 0),1, make_float3(0, 0, 0)), data,2);
	// IntersectSphere(ray, &best_hit,Sphere(make_float3(-2, 1, 0), 1, make_float3(1, 0, 0), make_float3(0, 0, 0),1, make_float3(0, 0, 0)), data, 3);
	return best_hit;
}

__device__ float3 total(float3* es, float3* as,int index)
{
	auto a = make_float3(0, 0, 0);
	for (int i=0;i<index;i++)
	{
		auto e = es[i];
		for (int j = 0; j < i;j++)e *= as[i];
		a += e;
	}
}

__device__ float3 Shade(Ray& ray, SurfaceHitRecord& hit, float3& factor, int depth, const RTDeviceData data,bool& _break)
{
	// auto c = make_float3(0, 0, 0);
	if (hit.t < 99999)
	{
		float3 random_in_unit_sphere;
		do random_in_unit_sphere = 2.0 * make_float3(data.GetRandom(), data.GetRandom(), data.GetRandom()) - make_float3(1, 1, 1);
		while (SquaredLength(random_in_unit_sphere) >= 1.0);
		auto scattered = Ray();
		float3 attenuation;
		auto emitted = hit.mat_ptr->emitted(hit.uv.x, hit.uv.y, hit.p);
		if (depth < 8 &&hit.mat_ptr->scatter(ray, hit, attenuation, scattered, random_in_unit_sphere, data))
		{
			auto e = emitted * factor;
			factor *= attenuation;
			ray = scattered;
			return make_float3(0, 0, 0);
		}
		else
		{
			_break = true;
			return emitted*factor;
		}
	}//total(es,as,depth)+
	else return factor * data.SampleTexture(0, atan2(ray.direction.x, -ray.direction.z) / -M_PI * 0.5f, acos(ray.direction.y) / -M_PI);

}


__global__ void IPRSampler(const int width, const int height, const int seed, const int spp,int Sampled, int mst, int root, float* output, hiprandState* const rngStates, Camera* camera,RTHostData host_data)
{
	const auto tidx = blockIdx.x * blockDim.x + threadIdx.x;
	const auto tidy = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * 16 + threadIdx.x,
	          y = blockIdx.y * 16 + threadIdx.y;

	if(host_data.quick)
	{
		if (!(x % PREVIEW_PIXEL_SIZE == 0 && y % PREVIEW_PIXEL_SIZE == 0)){
			const auto i = width * 4 * (y)+(x) * 4;
			output[i] =0;
			output[1 + i] = 0;
			output[2 + i] = 0;
			output[3 + i] = 1;
			return;
		}
	}


	hiprand_init((seed + tidx + width * tidy)*Sampled, 0, 0, &rngStates[tidx]);
	auto data = RTDeviceData(rngStates, tidx, Sampled,make_float2(x,y));
	data.Materials = host_data.Materials;
	data.Textures = host_data.Textures;
	data.bvh = host_data.bvh;
	data.ground = host_data.ground;


	auto color = make_float3(0, 0, 0);
	auto result = make_float3(0, 0, 0);

	
	//Main Sampling
	for (auto j = 0; j < spp; j++)
	{
		const auto u = float(data.GetRandom() + x) / float(width);
		const auto v = float(data.GetRandom() + y) / float(height);
		auto ray = CreateCameraRay(camera, u, v);
		auto factor = make_float3(1, 1, 1);

		for (auto i = 0; i < mst; i++)
		{
			auto hit = Trace(ray,data);
			auto _break = false;
			result =result+Shade(ray, hit, factor, i,data, _break);
		
			if (_break) break;
			
		}
		color =color+ result;
	}


	//Set color to buffer.
	const auto i = width * 4 * y + x * 4;
	if(Sampled==spp)
	{
		output[i] = color.x;
		output[1 + i]= color.y;
		output[2 + i] = color.z;
		output[3 + i] = spp;
	}
	else
	{
		output[i] += color.x;
		output[1 + i] += color.y;
		output[2 + i] += color.z;
		output[3 + i] += spp;
	}

}
