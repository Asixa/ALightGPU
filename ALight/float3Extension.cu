#include "hip/hip_runtime.h"
#include "float3Extension.h"
#include <vector_functions.hpp>
#include "math.h"
__host__ __device__ float3 operator+(const float3& lhs, const float3& rhs)
{
	return make_float3(lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z);
}

__host__ __device__ float3 operator+(const float3& v, const float& t)
{
	return make_float3(v.x + t, v.y + t, v.z + t);
}

__host__ __device__ float3 operator-(const float3& lhs, const float3& rhs)
{
	return make_float3(lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z);
}

__host__ __device__ float3 operator*(const float3& lhs, const float3& rhs)
{
	return make_float3(lhs.x * rhs.x, lhs.y * rhs.y, lhs.z * rhs.z);
}

__host__ __device__ float3 operator*(float t, const float3& v)
{
	return make_float3(v.x * t, v.y * t, v.z *t);
}

__host__ __device__ float3 operator*(const float3& v, float t)
{
	return make_float3(v.x * t, v.y * t, v.z * t);
}

__host__ __device__ float3 operator/(const float3& lhs, const float3& rhs)
{
	return make_float3(lhs.x / rhs.x, lhs.y / rhs.y, lhs.z / rhs.z);
}

__host__ __device__ float3 operator/(float3 v, float t)
{
	return make_float3(v.x / t, v.y / t, v.z / t);
}

__host__ __device__ float Dot(const float3& v1, const float3& v2)
{
	return v1.x* v2.x + v1.y * v2.y + v1.z * v2.z;
}

__host__ __device__ float3 Cross(const float3& v1, const float3& v2)
{
	return make_float3(v1.y * v2.z - v1.z * v2.y, (-(v1.x * v2.z - v1.z * v2.x)),
		(v1.x * v2.y - v1.y * v2.x));
}

__host__ __device__ float3 Cross2(const float3& lhs, const float3& rhs)
{
	return make_float3(lhs.y * rhs.z - lhs.z * rhs.y,
		lhs.z * rhs.x - lhs.x * rhs.z,
		lhs.x * rhs.y - lhs.y * rhs.x);
}

__host__ __device__ float Length(float3 f)
{
	return sqrt(f.x * f.x + f.y * f.y + f.z * f.z);
}

__host__ __device__ float SquaredLength(float3 f)
{
	return f.x * f.x + f.y * f.y + f.z * f.z;
}

__host__ __device__ bool IsZero(float3 f)
{
	return f.x == 0 && f.y == 0 && f.z == 0;
}

__host__ __device__ float3 UnitVector(float3 v)
{
	return v / Length(v);
}



__host__ __device__ float Distance(float3 a, float3 b)
{
	return Length(a - b);
}

__host__ __device__ void MakeUnitVector(float3* f)
{
	const float k = 1.0 / sqrt(f->x * f->x + f->y * f->y + f->z * f->z);
	f->x *= k; f->y *= k; f->z *= k;
}

__host__ __device__ float3 Reflect(float3 vin, float3 normal)
{
	return vin - 2 * Dot(vin, normal) * normal;
}

__host__ __device__ float3 Min(float3 a, float3 b)
{
	if (Length(a) < Length(b))return a;
	else return b;
}

__host__ __device__ float3 operator-(float3& a)
{
	return make_float3(-a.x, -a.y, -a.z);
}

__host__ __device__ float3 operator-(const float3& a)
{
	return make_float3(-a.x, -a.y, -a.z);
}

void Set(float3& f, float a, float b, float c)
{
	f.x = a;
	f.y = b;
	f.z = c;
}

void Set(float3& f, float a)
{
	f.x = a;
	f.y = a;
	f.z = a;
}
